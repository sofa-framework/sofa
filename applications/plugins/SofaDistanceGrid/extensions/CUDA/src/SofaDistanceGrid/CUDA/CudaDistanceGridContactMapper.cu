#include "hip/hip_runtime.h"
﻿/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <hip/hip_vector_types.h>
#include <sofa/gpu/cuda/mycuda.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void RigidContactMapperCuda3f_setPoints2(unsigned int size, unsigned int nbTests, unsigned int maxPoints, const void* tests, const void* contacts, void* map);
}

struct /*__align__(16)*/ GPUContact
{
    int p1;
    float3 p2;
    float distance;
    float3 normal;
};

struct /*__align__(8)*/ GPUTestEntry
{
    int firstIndex;
    int curSize;
    int maxSize;
    int newIndex;
    int elem1,elem2;
};

__shared__ GPUTestEntry curTestEntry;

__global__ void RigidContactMapperCuda3f_setPoints2_kernel(const GPUTestEntry* tests, const GPUContact* contacts, float3* map)
{
    if (threadIdx.x == 0)
        curTestEntry = tests[blockIdx.x];

    __syncthreads();

    GPUContact c = contacts[curTestEntry.firstIndex + threadIdx.x];
    if (threadIdx.x < curTestEntry.curSize)
    {
        map[curTestEntry.newIndex + threadIdx.x] = c.p2;
    }
}

void RigidContactMapperCuda3f_setPoints2(unsigned int size, unsigned int nbTests, unsigned int maxPoints, const void* tests, const void* contacts, void* map)
{
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    {RigidContactMapperCuda3f_setPoints2_kernel<<< grid, threads >>>((const GPUTestEntry*)tests, (GPUContact*)contacts, (float3*)map); mycudaDebugError("RigidContactMapperCuda3f_setPoints2_kernel");}
}

#if defined(__cplusplus) && CUDA_VERSION < 2000
}
}
}
#endif
