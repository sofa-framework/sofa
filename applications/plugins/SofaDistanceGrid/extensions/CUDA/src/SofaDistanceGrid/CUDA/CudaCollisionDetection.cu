#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include <sofa/gpu/cuda/mycuda.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void CudaCollisionDetection_runTests(unsigned int nbTests, unsigned int maxPoints, const void* tests, void* nresults);
}

struct /*__align__(16)*/ GPUContactPoint
{
    CudaVec3<float> p;
    int elem;
};

struct /*__align__(16)*/ GPUContact
{
//    int p1;
//    CudaVec3<float> p2;
    float distance;
    CudaVec3<float> normal;
};

struct GPUTest
{
    GPUContact* result;
    GPUContactPoint* result1;
    GPUContactPoint* result2;
    const CudaVec3<float>* points;
    const float* radius;
    const float* grid;
    //matrix3<float> rotation;
    CudaVec3<float> rotation_x,rotation_y,rotation_z;
    CudaVec3<float> translation;
    float margin;
    int nbPoints;
    int gridnx, gridny, gridnz;
    CudaVec3<float> gridbbmin, gridbbmax;
    CudaVec3<float> gridp0, gridinvdp;
};

struct GPUDeformedCube
{
    int elem;
    int ix,iy,iz;
    int points0, nbp;
    CudaVec3<float> initP0, invDP;
};

struct GPUDeformedCubeState
{
    CudaVec4<float> faces[6];
    CudaVec3<float> C0, Dx, Dy, Dz, Dxy, Dxz, Dyz, Dxyz;
    CudaVec3<float> center, radius;
};

struct GPUDeformedCubeBSphere
{
    CudaVec3<float> center;
    float radius;
};

struct GPUTestFFD
{
    GPUContact* result;
    GPUContactPoint* result1;
    GPUContactPoint* result2;
    const CudaVec3<float>* points;
    const float* radius;
    const float* grid;
    const GPUDeformedCube* ffdCubes;
    GPUDeformedCubeState* ffdState;
    GPUDeformedCubeBSphere* ffdBSphere;
    float margin;
    int nbPoints;
    int nbCubes;
    int gridnx, gridny, gridnz;
    CudaVec3<float> gridbbmin, gridbbmax;
    CudaVec3<float> gridp0, gridinvdp;
};

//////////////////////
// GPU-side methods //
//////////////////////

__shared__ GPUTest curTest;

__global__ void CudaCollisionDetection_runTests_kernel(const GPUTest* tests, int* nresults)
{
    if (threadIdx.x == 0)
        curTest = tests[blockIdx.x];


    __syncthreads();

    //! Dynamically allocated shared memory to compact results
    extern  __shared__  int scan[];

    CudaVec3<float> p0,p;
    float distance;
    CudaVec3<float> grad = CudaVec3<float>::make(0,0,0);
    //CudaVec3<float> normal;
    int n = 0;
    if (threadIdx.x < curTest.nbPoints)
    {
        p0 = curTest.points[threadIdx.x];
        //p = curTest.rotation * p;
        p = CudaVec3<float>::make(dot(curTest.rotation_x, p0), dot(curTest.rotation_y, p0), dot(curTest.rotation_z, p0));
        p += curTest.translation;

        CudaVec3<float> coefs = mul(p-curTest.gridp0, curTest.gridinvdp);
        int x = __float2int_rd(coefs.x);
        int y = __float2int_rd(coefs.y);
        int z = __float2int_rd(coefs.z);
        if ((unsigned)x < curTest.gridnx-1
            && (unsigned)y < curTest.gridny-1
            && (unsigned)z < curTest.gridnz-1)
        {
            int nx = curTest.gridnx;
            int nxny = nx*curTest.gridny;
            coefs.x -= __int2float_rd(x);
            coefs.y -= __int2float_rd(y);
            coefs.z -= __int2float_rd(z);
            const float* gval = curTest.grid + (x+nx*y+nxny*z);
            float d000      = gval[0        ];
            float d100_d000 = gval[1        ] - d000;
            float d010      = gval[  nx     ];
            float d110_d010 = gval[1+nx     ] - d010;
            float d001      = gval[     nxny];
            float d101_d001 = gval[1   +nxny] - d001;
            float d011      = gval[  nx+nxny];
            float d111_d011 = gval[1+nx+nxny] - d011;
            float dx00      = d000 + (d100_d000)*coefs.x;
            float dx10_dx00 = d010 + (d110_d010)*coefs.x - dx00;
            float dx01      = d001 + (d101_d001)*coefs.x;
            float dx11_dx01 = d011 + (d111_d011)*coefs.x - dx01;
            float dy0       = dx00 + (dx10_dx00)*coefs.y;
            float dy1_dy0   = dx01 + (dx11_dx01)*coefs.y - dy0;
            distance = dy0 + (dy1_dy0)*coefs.z;
            float r = 0;
            if (curTest.radius)
            {
                r = curTest.radius[threadIdx.x];
            }
            if (distance < curTest.margin+r)
            {
                n = 1;
                grad.z = dy1_dy0;
                grad.y = (dx10_dx00) + ((dx11_dx01)-(dx10_dx00))*coefs.z;
                dy0     = d100_d000 + (d110_d010 - d100_d000)*coefs.y;
                dy1_dy0 = d101_d001 + (d111_d011 - d101_d001)*coefs.y - dy0;
                grad.x = dy0 + (dy1_dy0)*coefs.z;
                grad *= invnorm(grad);
                //normal = grad;
                p -= grad*distance;
                //distance -= r;
                distance = r;
                //grad = CudaVec3<float>::make(0,1,1);
            }
        }
    }

    scan[threadIdx.x] = n;

    for (int i=1; i<curTest.nbPoints; i<<=1)
    {
        __syncthreads();
        if (threadIdx.x>=i)
            scan[threadIdx.x] = scan[threadIdx.x] + scan[threadIdx.x - i];
    }

    if (n)
    {
        int i = scan[threadIdx.x]-1;
        GPUContact c;
        //c.p1 = threadIdx.x;
        //c.p2 = p;
        c.distance = distance;
        c.normal = -grad;
        //c.normal = normal; //CudaVec3<float>::make(-grad.x,-grad.y,-grad.z); //-grad;
        curTest.result[i] = c;
        //curTest.result[scan[threadIdx.x]-1].p1 = threadIdx.x;
        //curTest.result[scan[threadIdx.x]-1].p2 = p;
        //curTest.result[scan[threadIdx.x]-1].distance = distance;
        //curTest.result[scan[threadIdx.x]-1].normal = normal;
        GPUContactPoint cp1;
        cp1.elem = threadIdx.x;
        cp1.p = p0;
        curTest.result1[i] = cp1;
        GPUContactPoint cp2;
        cp2.elem = 0;
        cp2.p = p;
        curTest.result2[i] = cp2;
    }
    if (threadIdx.x == curTest.nbPoints-1)
        nresults[blockIdx.x] = scan[curTest.nbPoints-1];
}

//////////////////////
// CPU-side methods //
//////////////////////

void CudaCollisionDetection_runTests(unsigned int nbTests, unsigned int maxPoints, const void* tests, void* nresults)
{
    sofa::gpu::cuda::mycudaPrintf("sizeof(GPUTest)=%d\nsizeof(GPUContact)=%d\nsizeof(matrix3<float>)=%d\n",sizeof(GPUTest),sizeof(GPUContact),sizeof(matrix3<float>));
    const GPUTest* gputests = (const GPUTest*)tests;
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    {CudaCollisionDetection_runTests_kernel<<< grid, threads, threads.x*sizeof(int) >>>(gputests, (int*)nresults); mycudaDebugError("CudaCollisionDetection_runTests_kernel");}

}

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
