#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void SubsetContactMapperCuda3f_setPoints1(unsigned int size, unsigned int nbTests, unsigned int maxPoints, unsigned int nbPointsPerElem, const void* tests, const void* contacts, void* map);
}

struct /*__align__(16)*/ GPUContact
{
    int p1;
    float3 p2;
    float distance;
    float3 normal;
};

struct /*__align__(8)*/ GPUTestEntry
{
    int firstIndex;
    int curSize;
    int maxSize;
    int newIndex;
    int elem1,elem2;
};

//////////////////////
// GPU-side methods //
//////////////////////

__shared__ GPUTestEntry curTestEntry;

__global__ void SubsetContactMapperCuda3f_setPoints1_kernel(unsigned int nbPointsPerElem, const GPUTestEntry* tests, const GPUContact* contacts, int* map)
{
    if (threadIdx.x == 0)
        curTestEntry = tests[blockIdx.x];

    __syncthreads();

    GPUContact c = contacts[curTestEntry.firstIndex + threadIdx.x];
    if (threadIdx.x < curTestEntry.curSize)
    {
        map[curTestEntry.newIndex + threadIdx.x] = curTestEntry.elem1 * nbPointsPerElem + c.p1;
    }
}


//////////////////////
// CPU-side methods //
//////////////////////

void SubsetContactMapperCuda3f_setPoints1(unsigned int size, unsigned int nbTests, unsigned int maxPoints, unsigned int nbPointsPerElem, const void* tests, const void* contacts, void* map)
{
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    {SubsetContactMapperCuda3f_setPoints1_kernel<<< grid, threads >>>(nbPointsPerElem, (const GPUTestEntry*)tests, (GPUContact*)contacts, (int*)map); mycudaDebugError("SubsetContactMapperCuda3f_setPoints1_kernel");}

}

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
