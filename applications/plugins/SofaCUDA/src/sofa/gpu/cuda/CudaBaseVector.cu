#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include "mycuda.h"
#include <hip/hip_runtime.h>

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

using namespace sofa::gpu::cuda;

extern "C"
{
    void SOFA_GPU_CUDA_API copy_vectorf(int dim,const void * a, void * b);
    void SOFA_GPU_CUDA_API vector_vector_peqf(int dim,float f,const void * a,void * b);
    void SOFA_GPU_CUDA_API sub_vector_vectorf(int dim,const void * a, const void * b, void * r);
    void SOFA_GPU_CUDA_API permute_vectorf(int dim,const void * a, const void * perm, void * b);

#ifdef SOFA_GPU_CUDA_DOUBLE
    void SOFA_GPU_CUDA_API copy_vectord(int dim,const void * a, void * b);
    void SOFA_GPU_CUDA_API vector_vector_peqd(int dim,double f,const void * a,void * b);
    void SOFA_GPU_CUDA_API sub_vector_vectord(int dim,const void * a, const void * b, void * r);
    void SOFA_GPU_CUDA_API permute_vectord(int dim,const void * a, const void * perm, void * b);
#endif
}


template<class real>
__global__ void Cuda_CopyVector_kernel(int dim, const real * a, real * b)
{
    int ti = blockIdx.x * BSIZE + threadIdx.x;
    if (ti >= dim) return;
    b[ti] = a[ti];
}

void SOFA_GPU_CUDA_API copy_vectorf(int dim,const void * a, void * b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {Cuda_CopyVector_kernel<float><<< grid, threads >>>(dim,(const float *) a,(float *) b); mycudaDebugError("Cuda_CopyVector_kernel<float>");}
}

#ifdef SOFA_GPU_CUDA_DOUBLE
void SOFA_GPU_CUDA_API copy_vectord(int dim,const void * a, void * b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {Cuda_CopyVector_kernel<double><<< grid, threads >>>(dim,(const double *) a,(double *) b); mycudaDebugError("Cuda_CopyVector_kernel<double>");}
}
#endif

template<class real>
__global__ void Cuda_vector_vector_peq_kernel(int dim,real f, const real * a, real * b)
{
    int ti = blockIdx.x * BSIZE + threadIdx.x;
    if (ti >= dim) return;
    b[ti] += a[ti]*f;
}

void vector_vector_peqf(int dim,float f,const void * a,void * b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {Cuda_vector_vector_peq_kernel<float><<< grid, threads >>>(dim,f,(const float *) a,(float *) b); mycudaDebugError("Cuda_vector_vector_peq_kernel<float>");}
}

#ifdef SOFA_GPU_CUDA_DOUBLE
void SOFA_GPU_CUDA_API vector_vector_peqd(int dim,double f,const void * a,void * b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {Cuda_vector_vector_peq_kernel<double><<< grid, threads >>>(dim,f,(const double *) a,(double *) b); mycudaDebugError("Cuda_vector_vector_peq_kernel<double>");}
}
#endif


template<class real>
__global__ void Cuda_sub_vector_kernel(int dim,const real * a, const real * b, real * r)
{
    int ti = blockIdx.x * BSIZE + threadIdx.x;
    if (ti >= dim) return;
    r[ti] = a[ti] - b[ti];
}

void sub_vector_vectorf(int dim,const void * a, const void * b, void * r)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {Cuda_sub_vector_kernel<float><<< grid, threads >>>(dim,(const float *) a,(const float *) b,(float *) r); mycudaDebugError("Cuda_sub_vector_kernel<float>");}
}

#ifdef SOFA_GPU_CUDA_DOUBLE
void SOFA_GPU_CUDA_API sub_vector_vectord(int dim,const void * a, const void * b, void * r)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {Cuda_sub_vector_kernel<double><<< grid, threads >>>(dim,(const double *) a,(const double *) b,(double *) r); mycudaDebugError("Cuda_sub_vector_kernel<double>");}
}
#endif


template<class real>
__global__ void permute_vector_kernel(int dim,const real * a, const int * perm, real * b)
{
    int ti = blockIdx.x * BSIZE + threadIdx.x;
    if (ti >= dim) return;
    b[ti] = a[perm[ti]];
}

void permute_vectorf(int dim,const void * a, const void * perm, void * b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {permute_vector_kernel<float><<< grid, threads >>>(dim,(const float *) a,(const int *) perm, (float *) b); mycudaDebugError("Cuda_sub_vector_kernel<float>");}
}

#ifdef SOFA_GPU_CUDA_DOUBLE
void SOFA_GPU_CUDA_API permute_vectord(int dim,const void * a, const void * perm, void * b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((dim+BSIZE-1)/BSIZE,1);

    {permute_vector_kernel<double><<< grid, threads >>>(dim,(const double *) a,(const int *) perm,(double *) b); mycudaDebugError("Cuda_sub_vector_kernel<double>");}
}
#endif

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
