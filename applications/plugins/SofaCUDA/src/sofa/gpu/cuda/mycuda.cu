#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#ifdef _WIN32
#include <windows.h>
#endif

#include "mycuda.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>

hipDeviceProp_t mycudaDeviceProp;


#if defined(__cplusplus)

#define STRINGIFY(x) #x
#define _STR(x) STRINGIFY(x)
SOFA_PRAGMA_MESSAGE("__cplusplus value: " _STR(__cplusplus))

namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    int SOFA_GPU_CUDA_API mycudaGetMultiProcessorCount();
    void cuda_void_kernel();
}

bool cudaCheck(hipError_t err, const char* src="?")
{
    if (err == hipSuccess) return true;
    //fprintf(stderr, "CUDA: Error %d returned from %s.\n",(int)err,src);
    mycudaLogError(hipGetErrorString(err), src);
    return false;
}

bool cudaInitCalled = false;
int deviceCount = 0;

#ifdef SOFA_WITH_DEVTOOLS
__global__ void print_cuda_standard()
{
    /**
    * 199711L = C++98
    * 201103L = C++11
    * 201402L = C++14
    * 201703L = C++17
    * 202002L = C++20
    */
    printf("CUDA Standard: %ld\n", __cplusplus);
}
#endif

int mycudaInit(int device)
{
    if (cudaInitCalled) return 1;

#if defined(__cplusplus)
    mycudaPrintf("C++ standard = %ld", __cplusplus);
#endif

    cudaInitCalled = true;
    const hipError_t getDeviceCountError = hipGetDeviceCount(&deviceCount);
    if (getDeviceCountError != hipSuccess)
    {
        mycudaPrintfError("error returned from hipGetDeviceCount: %s", hipGetErrorString(getDeviceCountError));
        return 0;
    }
    mycudaPrintf("CUDA: %d device(s) found.\n", deviceCount);
    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t dev
#ifdef cudaDevicePropDontCare
            = cudaDevicePropDontCare
#endif
                    ;
        //memset(&dev,0,sizeof(dev));
        //dev.name=NULL;
        //dev.bytes=0;
        //dev.major=0;
        //dev.minor=0;
        cudaCheck(hipGetDeviceProperties(&dev,i),"hipGetDeviceProperties");

        size_t free, total;
        hipMemGetInfo(&free,&total);

#if CUDA_VERSION >= 2010
        mycudaPrintf("CUDA:  %d : \"%s\", %d/%d MB, %d cores at %.3f GHz, revision %d.%d",i,dev.name, free/(1024*1024), dev.totalGlobalMem/(1024*1024), dev.multiProcessorCount*8, dev.clockRate * 1e-6f, dev.major, dev.minor);
        if (dev.kernelExecTimeoutEnabled)
            mycudaPrintf(", timeout enabled", dev.kernelExecTimeoutEnabled);
        mycudaPrintf("\n");
#elif CUDA_VERSION >= 2000
        mycudaPrintf("CUDA:  %d : \"%s\", %d/%d MB, %d cores at %.3f GHz, revision %d.%d\n",i,dev.name, free/(1024*1024), dev.totalGlobalMem/(1024*1024), dev.multiProcessorCount*8, dev.clockRate * 1e-6f, dev.major, dev.minor);
#else //if CUDA_VERSION >= 1000
        mycudaPrintf("CUDA:  %d : \"%s\", %d/%d MB, cores at %.3f GHz, revision %d.%d\n",i,dev.name, free/(1024*1024), dev.totalGlobalMem/(1024*1024), dev.clockRate * 1e-6f, dev.major, dev.minor);
//#else
//		mycudaPrintf("CUDA:  %d : \"%s\", %d/%d MB, revision %d.%d\n",i,(dev.name==NULL?"":dev.name), free/(1024*1024), dev.bytes/(1024*1024), dev.major, dev.minor);
#endif
    }
    if (device==-1)
    {
        const char* var = mygetenv("CUDA_DEVICE");
        device = (var && *var) ? atoi(var):0;
    }
    if (device >= deviceCount)
    {
        mycudaPrintf("CUDA: Device %d not found.\n", device);
        return 0;
    }
    else
    {
        hipDeviceProp_t& dev = mycudaDeviceProp;
        cudaCheck(hipGetDeviceProperties(&dev,device));
        mycudaPrintf("CUDA: Using device %d : \"%s\"\n",device,dev.name);
        cudaCheck(hipSetDevice(device));
        mycudaPrivateInit(device);
    }


#if defined(SOFA_GPU_CUBLAS) && !defined(SOFA_GPU_CUBLAS_V2)
    cublasInit();
#endif

#ifdef SOFA_WITH_DEVTOOLS
    print_cuda_standard<<<1, 1>>>();
    cudaCheck(hipDeviceSynchronize());
#endif

    return 1;
}

int mycudaGetMultiProcessorCount()
{
    return mycudaDeviceProp.multiProcessorCount;
}

void mycudaMalloc(void **devPtr, size_t size,int /*d*/)
{
    if (!cudaInitCalled) mycudaInit();
    if (mycudaVerboseLevel>=LOG_INFO) mycudaPrintf("CUDA: malloc(%d).\n",size);
    cudaCheck(hipMalloc(devPtr, size),"hipMalloc");
    if (mycudaVerboseLevel>=LOG_TRACE) mycudaPrintf("CUDA: malloc(%d) -> 0x%x.\n",size, *devPtr);
}

void mycudaMallocPitch(void **devPtr, size_t* pitch, size_t width, size_t height)
{
    if (!cudaInitCalled) mycudaInit();
    if (mycudaVerboseLevel>=LOG_INFO) mycudaPrintf("CUDA: mallocPitch(%d,%d).\n",width,height);
    cudaCheck(hipMallocPitch(devPtr, pitch, width, height),"cudaMalloc2D");
    if (mycudaVerboseLevel>=LOG_TRACE) mycudaPrintf("CUDA: mallocPitch(%d,%d) -> 0x%x at pitch %d.\n",width,height, *devPtr, (int)*pitch);
}

void mycudaFree(void *devPtr,int /*d*/)
{
    if (mycudaVerboseLevel>=LOG_TRACE) mycudaPrintf("CUDA: free(0x%x).\n",devPtr);
    cudaCheck(hipFree(devPtr),"hipFree");
}

void mycudaMallocHost(void **hostPtr, size_t size)
{
    if (!cudaInitCalled) mycudaInit();
    if (mycudaVerboseLevel>=LOG_TRACE) mycudaPrintf("CUDA: mallocHost(%d).\n",size);
    cudaCheck(hipHostMalloc(hostPtr, size),"hipHostMalloc");
    if (mycudaVerboseLevel>=LOG_TRACE) mycudaPrintf("CUDA: mallocHost(%d) -> 0x%x.\n",size, *hostPtr);
}

void mycudaFreeHost(void *hostPtr)
{
    if (mycudaVerboseLevel>=LOG_TRACE) mycudaPrintf("CUDA: freeHost(0x%x).\n",hostPtr);
    cudaCheck(hipHostFree(hostPtr),"hipHostFree");
}

void mycudaMemcpyHostToDevice(void *dst, const void *src, size_t count,int /*d*/)
{
    //count = (count+3)&(size_t)-4;
    if (!cudaCheck(hipMemcpy(dst, src, count, hipMemcpyHostToDevice),"hipMemcpyHostToDevice"))
        mycudaPrintf("in mycudaMemcpyHostToDevice(0x%x, 0x%x, %d)\n",dst,src,count);

    if (mycudaVerboseLevel>=LOG_STACK_TRACE) displayStack("mycudaMemcpyHostToDevice");
}

void mycudaMemcpyDeviceToDevice(void *dst, const void *src, size_t count,int /*d*/		)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice),"hipMemcpyDeviceToDevice");
}

void mycudaMemcpyDeviceToHost(void *dst, const void *src, size_t count,int /*d*/)
{
    //count = (count+3)&(size_t)-4;
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost),"hipMemcpyDeviceToHost");

    if (mycudaVerboseLevel>=LOG_STACK_TRACE) displayStack("mycudaMemcpyDeviceToHost");
}

void mycudaMemcpyHostToDevice2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height)
{
    cudaCheck(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyHostToDevice),"cudaMemcpyHostToDevice2D");

    if (mycudaVerboseLevel>=LOG_STACK_TRACE) displayStack("mycudaMemcpyHostToDevice2D");
}

void mycudaMemcpyDeviceToDevice2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height)
{
    cudaCheck(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice),"cudaMemcpyDeviceToDevice2D");
}

void mycudaMemcpyDeviceToHost2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height)
{
    cudaCheck(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost),"cudaMemcpyDeviceToHost2D");

    if (mycudaVerboseLevel>=LOG_STACK_TRACE) displayStack("mycudaMemcpyDeviceToHost2D");
}

void mycudaMemset(void *devPtr, int val, size_t size,int d)
{
    cudaCheck(hipMemset(devPtr, val,size),"mycudaMemset");
}



void mycudaThreadSynchronize()
{
    if (!cudaInitCalled) return; // no need to synchronize if no-one used cuda yet

    hipDeviceSynchronize();
}

#if CUDA_VERSION >= 4000

void mycudaDeviceSynchronize()
{
	if (!cudaInitCalled) return;
	
	hipDeviceSynchronize();
}
#endif

void mycudaCheckError(const char* src)
{
    if (!cudaInitCalled) return; // no need to check errors if no-one used cuda yet
    hipDeviceSynchronize();
    cudaCheck(hipGetLastError(),src);
}

void mycudaGLRegisterBufferObject(int id)
{
    if (!cudaInitCalled) mycudaInit();
    mycudaPrintf("mycudaGLRegisterBufferObject %d\n",id);
    cudaCheck(cudaGLRegisterBufferObject((GLuint)id),"cudaGLRegisterBufferObject");
}

void mycudaGLUnregisterBufferObject(int id)
{
    cudaCheck(cudaGLUnregisterBufferObject((GLuint)id),"cudaGLUnregisterBufferObject");
}

void mycudaGLMapBufferObject(void** ptr, int id)
{
    cudaCheck(cudaGLMapBufferObject(ptr, (GLuint)id),"cudaGLMapBufferObject");
}

void mycudaGLUnmapBufferObject(int id)
{
    cudaCheck(cudaGLUnmapBufferObject((GLuint)id),"cudaGLUnmapBufferObject");
}

int mycudaGetnumDevices()
{
    if (!cudaInitCalled) mycudaInit();
    return deviceCount;
}

int mycudaGetBufferDevice()
{
    return 0;
}

__global__ void cuda_debug_kernel()
{
}

void cuda_void_kernel()
{
    mycudaPrintf("WARNING : cuda_void_kernel should only be used for debug\n");

    dim3 threads(1,1);
    dim3 grid(1,1);
    {cuda_debug_kernel<<< grid, threads >>>(); mycudaDebugError("cuda_debug_kernel");}
}

#ifdef SOFA_GPU_CUBLAS

hipblasHandle_t getCublasCtx()
{
    static hipblasHandle_t cublashandle = NULL;
    if (cublashandle==NULL)
    {
        hipblasStatus_t status = hipblasCreate(&cublashandle);
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
            mycudaPrintf("cublas Handle init failed\n");
        }
    }
    return cublashandle;
}

hipsparseHandle_t getCusparseCtx()
{
    static hipsparseHandle_t cusparsehandle = NULL;
    if (cusparsehandle==NULL)
    {
        hipsparseStatus_t status = hipsparseCreate(&cusparsehandle);
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
            mycudaPrintf("cusparse Handle init failed\n");
        }
    }
    return cusparsehandle;
}

static hipsparseMatDescr_t matdescGen=NULL;

hipsparseMatDescr_t getCusparseMatGeneralDescr()
{
    if (matdescGen==NULL)
    {
        hipsparseStatus_t status = hipsparseCreateMatDescr(&matdescGen);
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
            mycudaPrintf("Matrix descriptor init failed\n");
        }
        hipsparseSetMatIndexBase(matdescGen, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(matdescGen, HIPSPARSE_MATRIX_TYPE_GENERAL);
    }

    return matdescGen;
}

static hipsparseMatDescr_t matdescTriLower=NULL;

hipsparseMatDescr_t getCusparseMatTriangularLowerDescr()
{
    if (matdescTriLower==NULL)
    {
        hipsparseStatus_t status = hipsparseCreateMatDescr(&matdescTriLower);
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
            mycudaPrintf("Matrix descriptor init failed\n");
        }
        hipsparseSetMatType ( matdescTriLower, HIPSPARSE_MATRIX_TYPE_TRIANGULAR );
        hipsparseSetMatIndexBase ( matdescTriLower, HIPSPARSE_INDEX_BASE_ZERO );
        hipsparseSetMatDiagType ( matdescTriLower, HIPSPARSE_DIAG_TYPE_UNIT );
        hipsparseSetMatFillMode ( matdescTriLower, HIPSPARSE_FILL_MODE_LOWER );
    }

    return matdescTriLower;
}

static hipsparseMatDescr_t matdescTriUpper=NULL;

hipsparseMatDescr_t getCusparseMatTriangularUpperDescr()
{
    if (matdescTriUpper==NULL)
    {
        hipsparseStatus_t status = hipsparseCreateMatDescr(&matdescTriUpper);
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
            mycudaPrintf("Matrix descriptor init failed\n");
        }
        hipsparseSetMatType ( matdescTriUpper, HIPSPARSE_MATRIX_TYPE_TRIANGULAR );
        hipsparseSetMatIndexBase ( matdescTriUpper, HIPSPARSE_INDEX_BASE_ZERO );
        hipsparseSetMatDiagType ( matdescTriUpper, HIPSPARSE_DIAG_TYPE_UNIT );
        hipsparseSetMatFillMode ( matdescTriUpper, HIPSPARSE_FILL_MODE_UPPER );
    }

    return matdescTriUpper;
}

void SOFA_GPU_CUDA_API mycudaMemGetInfo(size_t * free,size_t * total) {
    hipMemGetInfo(free,total);
}

#endif //SOFA_GPU_CUBLAS

#if defined(__cplusplus)
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
