#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void RigidMappingCuda3f_apply(unsigned int size, const matrix3<float>& rotation, const CudaVec3<float>& translation, void* out, void* rotated, const void* in);
    void RigidMappingCuda3f_applyJ(unsigned int size, const CudaVec3<float>& v, const CudaVec3<float>& omega, void* out, const void* rotated);
    void RigidMappingCuda3f_applyJT(unsigned int size, unsigned int nbloc, void* out, const void* rotated, const void* in);
}

//////////////////////
// GPU-side methods //
//////////////////////

__global__ void RigidMappingCuda3f_apply_kernel(unsigned int size, CudaVec3<float> rotation_x, CudaVec3<float> rotation_y, CudaVec3<float> rotation_z, CudaVec3<float> translation, float* out, float* rotated, const float* in)
{
    int index0 = blockIdx.x * BSIZE; //blockDim.x;
    int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    int base = index0 * 3;
    in  += base;
    out += base;
    rotated += base;

    temp[index1        ] = in[index1        ];
    temp[index1+  BSIZE] = in[index1+  BSIZE];
    temp[index1+2*BSIZE] = in[index1+2*BSIZE];

    __syncthreads();

    int index3 = index1 * 3;
    const CudaVec3<float> p = CudaVec3<float>::make(temp[index3  ],temp[index3+1],temp[index3+2]);

    // rotated
    //p = rotation*p;

    temp[index3  ] = dot(rotation_x,p);
    temp[index3+1] = dot(rotation_y,p);
    temp[index3+2] = dot(rotation_z,p);

    __syncthreads();

    rotated[index1        ] = temp[index1        ];
    rotated[index1+  BSIZE] = temp[index1+  BSIZE];
    rotated[index1+2*BSIZE] = temp[index1+2*BSIZE];

    __syncthreads();

    temp[index3  ] += translation.x;
    temp[index3+1] += translation.y;
    temp[index3+2] += translation.z;

    __syncthreads();

    out[index1        ] = temp[index1        ];
    out[index1+  BSIZE] = temp[index1+  BSIZE];
    out[index1+2*BSIZE] = temp[index1+2*BSIZE];
}

__global__ void RigidMappingCuda3f_applyJ_kernel(unsigned int size, CudaVec3<float> v, CudaVec3<float> omega, float* out, const float* rotated)
{
    int index0 = blockIdx.x * BSIZE; //blockDim.x;
    int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    int base = index0 * 3;
    out += base;
    rotated += base;

    temp[index1        ] = rotated[index1        ];
    temp[index1+  BSIZE] = rotated[index1+  BSIZE];
    temp[index1+2*BSIZE] = rotated[index1+2*BSIZE];

    __syncthreads();

    int index3 = index1 * 3;
    CudaVec3<float> p = v - cross(CudaVec3<float>::make(temp[index3  ],temp[index3+1],temp[index3+2]),omega);

    temp[index3  ] = p.x;
    temp[index3+1] = p.y;
    temp[index3+2] = p.z;

    __syncthreads();

    out[index1        ] = temp[index1        ];
    out[index1+  BSIZE] = temp[index1+  BSIZE];
    out[index1+2*BSIZE] = temp[index1+2*BSIZE];
}

__global__ void RigidMappingCuda3f_applyJT_kernel(unsigned int size, unsigned int nbloc, float* out, const float* rotated, const float* in)
{
    int index0 = blockIdx.x * BSIZE; //blockDim.x;
    int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    CudaVec3<float> t = CudaVec3<float>::make(0.0f, 0.0f, 0.0f);
    CudaVec3<float> r = CudaVec3<float>::make(0.0f, 0.0f, 0.0f);

    int index3 = index1 * 3;

    while (index0 < size)
    {

        int base = index0 * 3;

        temp[index1        ] = in[base+index1        ];
        temp[index1+  BSIZE] = in[base+index1+  BSIZE];
        temp[index1+2*BSIZE] = in[base+index1+2*BSIZE];
        temp[index1+3*BSIZE] = rotated[base+index1        ];
        temp[index1+4*BSIZE] = rotated[base+index1+  BSIZE];
        temp[index1+5*BSIZE] = rotated[base+index1+2*BSIZE];

        __syncthreads();

        if (index0+index1 < size)
        {
            CudaVec3<float> v = CudaVec3<float>::make(temp[index3  ],temp[index3+1],temp[index3+2]);
            t += v;
            r += cross(CudaVec3<float>::make(temp[index3  +3*BSIZE],temp[index3+1+3*BSIZE],temp[index3+2+3*BSIZE]),v);
        }
        __syncthreads();
        index0 += nbloc * BSIZE;
    }

    temp[index3  ] = t.x;
    temp[index3+1] = t.y;
    temp[index3+2] = t.z;
    temp[index3  +3*BSIZE] = r.x;
    temp[index3+1+3*BSIZE] = r.y;
    temp[index3+2+3*BSIZE] = r.z;

    //__syncthreads();

    int offset = BSIZE/2;
    int offset3 = (BSIZE/2) * 3;

    while(offset>0)
    {
        //if (index1 >= offset && index1 < BSIZE)
        //	    temp[index1] = acc;
        __syncthreads();
        if (index1 < offset)
        {
            temp[index3  ]+=temp[index3+offset3  ];
            temp[index3+1]+=temp[index3+offset3+1];
            temp[index3+2]+=temp[index3+offset3+2];
            temp[index3  +3*BSIZE]+=temp[index3+offset3  +3*BSIZE];
            temp[index3+1+3*BSIZE]+=temp[index3+offset3+1+3*BSIZE];
            temp[index3+2+3*BSIZE]+=temp[index3+offset3+2+3*BSIZE];
        }
        offset >>= 1;
        offset3 >>= 1;
    }
    __syncthreads();
    if (index1 < 6)
    {
        out[blockIdx.x * 6 + index1] = temp[index1];
    }
}

//////////////////////
// CPU-side methods //
//////////////////////

void RigidMappingCuda3f_apply(unsigned int size, const matrix3<float>& rotation, const CudaVec3<float>& translation, void* out, void* rotated, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {RigidMappingCuda3f_apply_kernel<<< grid, threads, BSIZE*3*sizeof(float) >>>(size, rotation.x, rotation.y, rotation.z, translation, (float*)out, (float*)rotated, (const float*)in); mycudaDebugError("RigidMappingCuda3f_apply_kernel");}
}

void RigidMappingCuda3f_applyJ(unsigned int size, const CudaVec3<float>& v, const CudaVec3<float>& omega, void* out, const void* rotated)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {RigidMappingCuda3f_applyJ_kernel<<< grid, threads, BSIZE*3*sizeof(float) >>>(size, v, omega, (float*)out, (const float*)rotated); mycudaDebugError("RigidMappingCuda3f_applyJ_kernel");}
}

void RigidMappingCuda3f_applyJT(unsigned int size, unsigned int nbloc, void* out, const void* rotated, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid(nbloc,1);
    {RigidMappingCuda3f_applyJT_kernel<<< grid, threads, BSIZE*6*sizeof(float) >>>(size, nbloc, (float*)out, (const float*)rotated, (const float*)in); mycudaDebugError("RigidMappingCuda3f_applyJT_kernel");}
}

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
