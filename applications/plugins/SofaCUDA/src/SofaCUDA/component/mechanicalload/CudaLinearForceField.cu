#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include <sofa/gpu/cuda/CudaMathRigid.h>

#if defined(__cplusplus)
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif


extern "C"
{
    void LinearForceFieldCudaRigid3f_addForce(unsigned size, const void* indices, const void* forces, void* f);
#ifdef SOFA_GPU_CUDA_DOUBLE
    void LinearForceFieldCudaRigid3d_addForce(unsigned size, const void* indices, const void *forces, void* f);
#endif
}// extern "C"


//////////////////////
// GPU-side methods //
//////////////////////

template<class real>
__global__ void LinearForceFieldCudaRigid3t_addForce_kernel(unsigned size, const int* indices, real fx, real fy, real fz, real frx, real fry, real frz, CudaRigidDeriv3<real>* f)
{
    int index = blockIdx.x * BSIZE+threadIdx.x;

    CudaRigidDeriv3<real> force = CudaRigidDeriv3<real>::make(fx, fy, fz, frx, fry, frz);
    if (index < size)
    {
        f[indices[index]] += force;
    }
}// addForce_kernel

//////////////////////
// CPU-side methods //
//////////////////////

void LinearForceFieldCudaRigid3f_addForce(unsigned size, const void* indices, const void* forces, void* f)
{
//	float* fv = (float*)forces;
// 	mycudaPrintf("Add : %f %f %f %f %f %f\n", fv[0], fv[1], fv[2], fv[3], fv[4], fv[5]);
    dim3 threads(BSIZE, 1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    LinearForceFieldCudaRigid3t_addForce_kernel<float>
    <<< grid, threads >>>
    (size, (const int*)indices, ((float*)forces)[0], ((float*)forces)[1], ((float*)forces)[2], ((float*)forces)[3], ((float*)forces)[4], ((float*)forces)[5], (CudaRigidDeriv3<float>*)f);
}// LinearForceFieldCudaRigid3f_addForce

#ifdef SOFA_GPU_CUDA_DOUBLE
void LinearForceFieldCudaRigid3d_addForce(unsigned size, const void* indices, const void* forces, void* f)
{
    dim3 threads(BSIZE, 1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    LinearForceFieldCudaRigid3t_addForce_kernel<double>
    <<< grid, threads >>>
    (size, (const int*)indices, ((double*)f)[0], ((double*)f)[1], ((double*)f)[2], ((double*)f)[3], ((double*)f)[4], ((double*)f)[5], (CudaRigidDeriv3<double>*)f);
}// LinearForceFieldCudaRigid3f_addForce
#endif // SOFA_GPU_CUDA_DOUBLE

#if defined(__cplusplus)
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
