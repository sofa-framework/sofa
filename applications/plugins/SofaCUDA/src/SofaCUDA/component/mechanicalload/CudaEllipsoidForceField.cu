#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

struct GPUEllipsoid
{
    CudaVec3<float> center;
    CudaVec3<float> inv_r2;
    float stiffness;
    float damping;
};

extern "C"
{
    void EllipsoidForceFieldCuda3f_addForce(unsigned int size, GPUEllipsoid* ellipsoid, float* tmp, void* f, const void* x, const void* v);
    void EllipsoidForceFieldCuda3f_addDForce(unsigned int size, GPUEllipsoid* ellipsoid, const float* tmp, void* f, const void* dx, double factor); //, const void* dfdx);

    void EllipsoidForceFieldCuda3f1_addForce(unsigned int size, GPUEllipsoid* ellipsoid, float* tmp, void* f, const void* x, const void* v);
    void EllipsoidForceFieldCuda3f1_addDForce(unsigned int size, GPUEllipsoid* ellipsoid, const float* tmp, void* f, const void* dx, double factor); //, const void* dfdx);

    int EllipsoidForceFieldCuda3f_getNTmp();
}

//////////////////////
// GPU-side methods //
//////////////////////

#define NTMP 10

int EllipsoidForceFieldCuda3f_getNTmp()
{
    return NTMP;
}

__global__ void EllipsoidForceFieldCuda3f_addForce_kernel(int size, GPUEllipsoid ellipsoid, float* tmp, float* f, const float* x, const float* v)
{
    int index0 = blockIdx.x * BSIZE;
    int index0_3 = index0 * 3;

    tmp += index0*NTMP;
    f += index0_3;
    x += index0_3;
    v += index0_3;

    int index = threadIdx.x;
    int index_3 = index * 3;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    temp[index        ] = x[index        ];
    temp[index+  BSIZE] = x[index+  BSIZE];
    temp[index+2*BSIZE] = x[index+2*BSIZE];

    __syncthreads();

    CudaVec3<float> dp = CudaVec3<float>::make(temp[index_3  ], temp[index_3+1], temp[index_3+2]) - ellipsoid.center;

    __syncthreads();

    temp[index        ] = v[index        ];
    temp[index+  BSIZE] = v[index+  BSIZE];
    temp[index+2*BSIZE] = v[index+2*BSIZE];

    __syncthreads();

    CudaVec3<float> vi = CudaVec3<float>::make(temp[index_3  ], temp[index_3+1], temp[index_3+2]);
    CudaVec3<float> force = CudaVec3<float>::make(0,0,0);

    //float d2 = dot(dp,dp);
    CudaVec3<float> grad = CudaVec3<float>::make(dp.x*ellipsoid.inv_r2.x, dp.y*ellipsoid.inv_r2.y, dp.z*ellipsoid.inv_r2.z);
    //float norm2 = dp.x*dp.x*ellipsoid.inv_r2.x + dp.y*dp.y*ellipsoid.inv_r2.y + dp.z*dp.z*ellipsoid.inv_r2.z;
    float norm2 = dp.x*grad.x + dp.y*grad.y + dp.z*grad.z;
    //Real d = (norm2-1)*s2;
    CudaVec3<float> mx,my,mz;
    float d = (norm2-1)*ellipsoid.stiffness;
    tmp[index+0*BSIZE] = d;
    if (d<0)
    {
        //float norm = sqrt(norm2);
        float inv_norm = rsqrtf(norm2);
        float norm = 1.0f/inv_norm;
        const float stiffabs = abs(ellipsoid.stiffness);
        float v = norm-1;
        //for (int j=0; j<N; j++) grad[j] = dp[j]*inv_r2[j];
        float gnorm2 = dot(grad,grad);
        float inv_gnorm = rsqrtf(gnorm2);
        //grad /= gnorm; //.normalize();
        float forceIntensity = -stiffabs*v*inv_gnorm;
        float dampingIntensity = ellipsoid.damping*abs(v);
        force = grad*forceIntensity - vi*dampingIntensity;
        float fact1 = -stiffabs*inv_norm*inv_gnorm;
        float fact2 = -stiffabs*v*inv_gnorm;
        float fact3 = fact2*inv_gnorm; //-stiffabs*v / gnorm2;
        mx.x = grad.x*(grad.x*(fact1+fact3*ellipsoid.inv_r2.x)) + fact2*ellipsoid.inv_r2.x;
        mx.y = grad.x*(grad.y*(fact1+fact3*ellipsoid.inv_r2.y));
        mx.z = grad.x*(grad.z*(fact1+fact3*ellipsoid.inv_r2.z));
        my.x = grad.y*(grad.x*(fact1+fact3*ellipsoid.inv_r2.x));
        my.y = grad.y*(grad.y*(fact1+fact3*ellipsoid.inv_r2.y)) + fact2*ellipsoid.inv_r2.y;
        my.z = grad.y*(grad.z*(fact1+fact3*ellipsoid.inv_r2.z));
        mz.x = grad.z*(grad.x*(fact1+fact3*ellipsoid.inv_r2.x));
        mz.y = grad.z*(grad.y*(fact1+fact3*ellipsoid.inv_r2.y));
        mz.z = grad.z*(grad.z*(fact1+fact3*ellipsoid.inv_r2.z)) + fact2*ellipsoid.inv_r2.z;
    }
    tmp[index+1*BSIZE] = mx.x;
    tmp[index+2*BSIZE] = mx.y;
    tmp[index+3*BSIZE] = mx.z;
    tmp[index+4*BSIZE] = my.x;
    tmp[index+5*BSIZE] = my.y;
    tmp[index+6*BSIZE] = my.z;
    tmp[index+7*BSIZE] = mz.x;
    tmp[index+8*BSIZE] = mz.y;
    tmp[index+9*BSIZE] = mz.z;

    __syncthreads();

    temp[index_3+0] = force.x;
    temp[index_3+1] = force.y;
    temp[index_3+2] = force.z;

    __syncthreads();

    f[index        ] += temp[index        ];
    f[index+  BSIZE] += temp[index+  BSIZE];
    f[index+2*BSIZE] += temp[index+2*BSIZE];
}

__global__ void EllipsoidForceFieldCuda3f1_addForce_kernel(int size, GPUEllipsoid ellipsoid, float* tmp, CudaVec4<float>* f, const CudaVec4<float>* x, const CudaVec4<float>* v)
{
    int index = blockIdx.x * BSIZE+threadIdx.x;
    tmp += blockIdx.x * BSIZE * NTMP;

    CudaVec4<float> temp = x[index];
    CudaVec3<float> dp = CudaVec3<float>::make(temp) - ellipsoid.center;
    //float d2 = dot(dp,dp);

    const CudaVec4<float> vi = v[index];
    CudaVec3<float> force = CudaVec3<float>::make(0,0,0);

    CudaVec3<float> grad = CudaVec3<float>::make(dp.x*ellipsoid.inv_r2.x, dp.y*ellipsoid.inv_r2.y, dp.z*ellipsoid.inv_r2.z);
    //float norm2 = dp.x*dp.x*ellipsoid.inv_r2.x + dp.y*dp.y*ellipsoid.inv_r2.y + dp.z*dp.z*ellipsoid.inv_r2.z;
    float norm2 = dp.x*grad.x + dp.y*grad.y + dp.z*grad.z;
    //float d = (norm2-1)*s2;
    CudaVec3<float> mx,my,mz;
    float d = (norm2-1)*ellipsoid.stiffness;
    tmp[threadIdx.x+0*BSIZE] = d;
    if (d<0)
    {
        //float norm = sqrt(norm2);
        float inv_norm = rsqrtf(norm2);
        float norm = 1.0f/inv_norm;
        const float stiffabs = abs(ellipsoid.stiffness);
        float v = norm-1;
        //for (int j=0; j<N; j++) grad[j] = dp[j]*inv_r2[j];
        float gnorm2 = dot(grad,grad);
        float inv_gnorm = rsqrtf(gnorm2);
        //grad /= gnorm; //.normalize();
        float forceIntensity = -stiffabs*v*inv_gnorm;
        float dampingIntensity = ellipsoid.damping*abs(v);
        force = grad*forceIntensity - CudaVec3<float>::make(vi)*dampingIntensity;
        float fact1 = -stiffabs*inv_norm*inv_gnorm;
        float fact2 = -stiffabs*v*inv_gnorm;
        float fact3 = fact2*inv_gnorm; //-stiffabs*v / gnorm2;
        mx.x = grad.x*(grad.x*(fact1+fact3*ellipsoid.inv_r2.x)) + fact2*ellipsoid.inv_r2.x;
        mx.y = grad.x*(grad.y*(fact1+fact3*ellipsoid.inv_r2.y));
        mx.z = grad.x*(grad.z*(fact1+fact3*ellipsoid.inv_r2.z));
        my.x = grad.y*(grad.x*(fact1+fact3*ellipsoid.inv_r2.x));
        my.y = grad.y*(grad.y*(fact1+fact3*ellipsoid.inv_r2.y)) + fact2*ellipsoid.inv_r2.y;
        my.z = grad.y*(grad.z*(fact1+fact3*ellipsoid.inv_r2.z));
        mz.x = grad.z*(grad.x*(fact1+fact3*ellipsoid.inv_r2.x));
        mz.y = grad.z*(grad.y*(fact1+fact3*ellipsoid.inv_r2.y));
        mz.z = grad.z*(grad.z*(fact1+fact3*ellipsoid.inv_r2.z)) + fact2*ellipsoid.inv_r2.z;
    }
    tmp[threadIdx.x+1*BSIZE] = mx.x;
    tmp[threadIdx.x+2*BSIZE] = mx.y;
    tmp[threadIdx.x+3*BSIZE] = mx.z;
    tmp[threadIdx.x+4*BSIZE] = my.x;
    tmp[threadIdx.x+5*BSIZE] = my.y;
    tmp[threadIdx.x+6*BSIZE] = my.z;
    tmp[threadIdx.x+7*BSIZE] = mz.x;
    tmp[threadIdx.x+8*BSIZE] = mz.y;
    tmp[threadIdx.x+9*BSIZE] = mz.z;

    temp = f[index];
    temp.x += force.x;
    temp.y += force.y;
    temp.z += force.z;
    f[index] = temp;
}

__global__ void EllipsoidForceFieldCuda3f_addDForce_kernel(int size, const float* tmp, float* df, const float* dx, float factor)
{
    int index0 = blockIdx.x * BSIZE;
    int index0_3 = index0 * 3;

    tmp += index0*NTMP;
    df += index0_3;
    dx += index0_3;

    int index = threadIdx.x;
    int index_3 = index * 3;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    temp[index        ] = dx[index        ];
    temp[index+  BSIZE] = dx[index+  BSIZE];
    temp[index+2*BSIZE] = dx[index+2*BSIZE];

    __syncthreads();

    const CudaVec3<float> dxi = CudaVec3<float>::make(temp[index_3  ], temp[index_3+1], temp[index_3+2]);
    float d = tmp[index+0*BSIZE];
    CudaVec3<float> mx,my,mz;
    mx.x = tmp[index+1*BSIZE];
    mx.y = tmp[index+2*BSIZE];
    mx.z = tmp[index+3*BSIZE];
    my.x = tmp[index+4*BSIZE];
    my.y = tmp[index+5*BSIZE];
    my.z = tmp[index+6*BSIZE];
    mz.x = tmp[index+7*BSIZE];
    mz.y = tmp[index+8*BSIZE];
    mz.z = tmp[index+9*BSIZE];

    CudaVec3<float> dforce = CudaVec3<float>::make(0,0,0);

    if (d<0)
    {
        dforce.x = dot(mx,dxi)*factor;
        dforce.y = dot(my,dxi)*factor;
        dforce.z = dot(mz,dxi)*factor;
    }

    __syncthreads();

    temp[index_3+0] = dforce.x;
    temp[index_3+1] = dforce.y;
    temp[index_3+2] = dforce.z;

    __syncthreads();

    df[index        ] += temp[index        ];
    df[index+  BSIZE] += temp[index+  BSIZE];
    df[index+2*BSIZE] += temp[index+2*BSIZE];
}

__global__ void EllipsoidForceFieldCuda3f1_addDForce_kernel(int size, const float* tmp, CudaVec4<float>* df, const CudaVec4<float>* dx, float factor)
{
    int index = blockIdx.x * BSIZE+threadIdx.x;
    tmp += blockIdx.x * BSIZE * NTMP;

    const CudaVec4<float> dxi = dx[index];
    float d = tmp[threadIdx.x+0*BSIZE];
    CudaVec3<float> mx,my,mz;
    mx.x = tmp[threadIdx.x+1*BSIZE];
    mx.y = tmp[threadIdx.x+2*BSIZE];
    mx.z = tmp[threadIdx.x+3*BSIZE];
    my.x = tmp[threadIdx.x+4*BSIZE];
    my.y = tmp[threadIdx.x+5*BSIZE];
    my.z = tmp[threadIdx.x+6*BSIZE];
    mz.x = tmp[threadIdx.x+7*BSIZE];
    mz.y = tmp[threadIdx.x+8*BSIZE];
    mz.z = tmp[threadIdx.x+9*BSIZE];

    CudaVec3<float> dforce = CudaVec3<float>::make(0,0,0);

    if (d<0)
    {
        dforce.x = dot(mx,CudaVec3<float>::make(dxi))*factor;
        dforce.y = dot(my,CudaVec3<float>::make(dxi))*factor;
        dforce.z = dot(mz,CudaVec3<float>::make(dxi))*factor;
    }

    CudaVec4<float> dfi = df[index];
    dfi.x += dforce.x;
    dfi.y += dforce.y;
    dfi.y += dforce.z;
    df[index] = dfi;
}

//////////////////////
// CPU-side methods //
//////////////////////

void EllipsoidForceFieldCuda3f_addForce(unsigned int size, GPUEllipsoid* ellipsoid, float* tmp, void* f, const void* x, const void* v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {EllipsoidForceFieldCuda3f_addForce_kernel<<< grid, threads, BSIZE*3*sizeof(float) >>>(size, *ellipsoid, tmp, (float*)f, (const float*)x, (const float*)v); mycudaDebugError("EllipsoidForceFieldCuda3f_addForce_kernel");}
}

void EllipsoidForceFieldCuda3f1_addForce(unsigned int size, GPUEllipsoid* ellipsoid, float* tmp, void* f, const void* x, const void* v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {EllipsoidForceFieldCuda3f1_addForce_kernel<<< grid, threads >>>(size, *ellipsoid, tmp, (CudaVec4<float>*)f, (const CudaVec4<float>*)x, (const CudaVec4<float>*)v); mycudaDebugError("EllipsoidForceFieldCuda3f1_addForce_kernel");}
}

void EllipsoidForceFieldCuda3f_addDForce(unsigned int size, GPUEllipsoid* ellipsoid, const float* tmp, void* df, const void* dx, double factor) //, const void* dfdx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {EllipsoidForceFieldCuda3f_addDForce_kernel<<< grid, threads, BSIZE*3*sizeof(float) >>>(size, /* *ellipsoid, */ tmp, (float*)df, (const float*)dx, (float)factor); mycudaDebugError("EllipsoidForceFieldCuda3f_addDForce_kernel");}
}

void EllipsoidForceFieldCuda3f1_addDForce(unsigned int size, GPUEllipsoid* ellipsoid, const float* tmp, void* df, const void* dx, double factor) //, const void* dfdx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {EllipsoidForceFieldCuda3f1_addDForce_kernel<<< grid, threads >>>(size, /* *ellipsoid, */ tmp, (CudaVec4<float>*)df, (const CudaVec4<float>*)dx, (float)factor); mycudaDebugError("EllipsoidForceFieldCuda3f1_addDForce_kernel");}
}

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
