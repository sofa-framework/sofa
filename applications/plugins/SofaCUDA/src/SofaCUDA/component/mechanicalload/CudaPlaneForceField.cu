#include "hip/hip_runtime.h"
/******************************************************************************
*                 SOFA, Simulation Open-Framework Architecture                *
*                    (c) 2006 INRIA, USTL, UJF, CNRS, MGH                     *
*                                                                             *
* This program is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This program is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this program. If not, see <http://www.gnu.org/licenses/>.        *
*******************************************************************************
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

template<class real>
class GPUPlane
{
public:
    //CudaVec3<real> normal;
    real normal_x, normal_y, normal_z;
    real d;
    real stiffness;
    real damping;
};

typedef GPUPlane<float> GPUPlane3f;
typedef GPUPlane<double> GPUPlane3d;

extern "C"
{
    void PlaneForceFieldCuda3f_addForce(unsigned int size, GPUPlane3f* plane, float* penetration, void* f, const void* x, const void* v);
    void PlaneForceFieldCuda3f_addDForce(unsigned int size, GPUPlane3f* plane, const float* penetration, void* f, const void* dx); //, const void* dfdx);

    void PlaneForceFieldCuda3f1_addForce(unsigned int size, GPUPlane3f* plane, float* penetration, void* f, const void* x, const void* v);
    void PlaneForceFieldCuda3f1_addDForce(unsigned int size, GPUPlane3f* plane, const float* penetration, void* f, const void* dx); //, const void* dfdx);

#ifdef SOFA_GPU_CUDA_DOUBLE

    void PlaneForceFieldCuda3d_addForce(unsigned int size, GPUPlane3d* plane, double* penetration, void* f, const void* x, const void* v);
    void PlaneForceFieldCuda3d_addDForce(unsigned int size, GPUPlane3d* plane, const double* penetration, void* f, const void* dx); //, const void* dfdx);

    void PlaneForceFieldCuda3d1_addForce(unsigned int size, GPUPlane3d* plane, double* penetration, void* f, const void* x, const void* v);
    void PlaneForceFieldCuda3d1_addDForce(unsigned int size, GPUPlane3d* plane, const double* penetration, void* f, const void* dx); //, const void* dfdx);

#endif // SOFA_GPU_CUDA_DOUBLE
}

//////////////////////
// GPU-side methods //
//////////////////////

template<class real>
__global__ void PlaneForceFieldCuda3t_addForce_kernel(int size, GPUPlane<real> plane, real* penetration, real* f, const real* x, const real* v)
{
    int index0 = blockIdx.x * BSIZE;
    int index0_3 = index0 * 3;

    penetration += index0;
    f += index0_3;
    x += index0_3;
    v += index0_3;

    int index = threadIdx.x;
    int index_3 = index * 3;

    //! Dynamically allocated shared memory to reorder global memory access
    __shared__  real temp[BSIZE*3];

    temp[index        ] = x[index        ];
    temp[index+  BSIZE] = x[index+  BSIZE];
    temp[index+2*BSIZE] = x[index+2*BSIZE];

    __syncthreads();

    CudaVec3<real> xi = CudaVec3<real>::make(temp[index_3  ], temp[index_3+1], temp[index_3+2]);
    real d = dot(xi,CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z))-plane.d;

    penetration[index] = d;

    __syncthreads();

    temp[index        ] = v[index        ];
    temp[index+  BSIZE] = v[index+  BSIZE];
    temp[index+2*BSIZE] = v[index+2*BSIZE];

    __syncthreads();

    CudaVec3<real> vi = CudaVec3<real>::make(temp[index_3  ], temp[index_3+1], temp[index_3+2]);
    CudaVec3<real> force = CudaVec3<real>::make(0,0,0);

    if (d<0)
    {
        real forceIntensity = -plane.stiffness*d;
        real dampingIntensity = -plane.damping*d;
        force = CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z)*forceIntensity - vi*dampingIntensity;
    }

    __syncthreads();

    temp[index        ] = f[index        ];
    temp[index+  BSIZE] = f[index+  BSIZE];
    temp[index+2*BSIZE] = f[index+2*BSIZE];

    __syncthreads();

    temp[index_3+0] += force.x;
    temp[index_3+1] += force.y;
    temp[index_3+2] += force.z;

    __syncthreads();

    f[index        ] = temp[index        ];
    f[index+  BSIZE] = temp[index+  BSIZE];
    f[index+2*BSIZE] = temp[index+2*BSIZE];
}

template<class real>
__global__ void PlaneForceFieldCuda3t1_addForce_kernel(int size, GPUPlane<real> plane, real* penetration, CudaVec4<real>* f, const CudaVec4<real>* x, const CudaVec4<real>* v)
{
    int index = blockIdx.x * BSIZE + threadIdx.x;

    CudaVec4<real> xi = x[index];
    real d = dot(CudaVec3<real>::make(xi),CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z))-plane.d;

    penetration[index] = d;

    CudaVec4<real> vi = v[index];
    CudaVec3<real> force = CudaVec3<real>::make(0,0,0);

    if (d<0)
    {
        real forceIntensity = -plane.stiffness*d;
        real dampingIntensity = -plane.damping*d;
        force = CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z)*forceIntensity - CudaVec3<real>::make(vi)*dampingIntensity;
    }

    CudaVec4<real> temp = f[index];
    temp.x += force.x;
    temp.y += force.y;
    temp.z += force.z;
    f[index] = temp;
}

template<class real>
__global__ void PlaneForceFieldCuda3t_addDForce_kernel(int size, GPUPlane<real> plane, const real* penetration, real* df, const real* dx)
{
    int index0 = blockIdx.x * BSIZE;
    int index0_3 = index0 * 3;

    penetration += index0;
    df += index0_3;
    dx += index0_3;

    int index = threadIdx.x;
    int index_3 = index * 3;

    //! Dynamically allocated shared memory to reorder global memory access
    __shared__  real temp[BSIZE*3];

    temp[index        ] = dx[index        ];
    temp[index+  BSIZE] = dx[index+  BSIZE];
    temp[index+2*BSIZE] = dx[index+2*BSIZE];

    __syncthreads();

    CudaVec3<real> dxi = CudaVec3<real>::make(temp[index_3  ], temp[index_3+1], temp[index_3+2]);
    real d = penetration[index];

    CudaVec3<real> dforce = CudaVec3<real>::make(0,0,0);

    if (d<0)
    {
        dforce = CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z) * (-plane.stiffness * dot(dxi, CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z)));
    }

    __syncthreads();

    temp[index        ] = df[index        ];
    temp[index+  BSIZE] = df[index+  BSIZE];
    temp[index+2*BSIZE] = df[index+2*BSIZE];

    __syncthreads();

    temp[index_3+0] += dforce.x;
    temp[index_3+1] += dforce.y;
    temp[index_3+2] += dforce.z;

    __syncthreads();

    df[index        ] = temp[index        ];
    df[index+  BSIZE] = temp[index+  BSIZE];
    df[index+2*BSIZE] = temp[index+2*BSIZE];
}

template<class real>
__global__ void PlaneForceFieldCuda3t1_addDForce_kernel(int size, GPUPlane<real> plane, const real* penetration, CudaVec4<real>* df, const CudaVec4<real>* dx)
{
    int index = blockIdx.x * BSIZE + threadIdx.x;

    CudaVec4<real> dxi = dx[index];
    real d = penetration[index];

    CudaVec3<real> dforce = CudaVec3<real>::make(0,0,0);

    if (d<0)
    {
        dforce = CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z) * (-plane.stiffness * dot(CudaVec3<real>::make(dxi), CudaVec3<real>::make(plane.normal_x,plane.normal_y,plane.normal_z)));
    }
    CudaVec4<real> dfi = df[index];
    dfi.x += dforce.x;
    dfi.y += dforce.y;
    dfi.y += dforce.z;
    df[index] = dfi;
}

//////////////////////
// CPU-side methods //
//////////////////////

void PlaneForceFieldCuda3f_addForce(unsigned int size, GPUPlane3f* plane, float* penetration, void* f, const void* x, const void* v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t_addForce_kernel<float><<< grid, threads >>>(size, *plane, penetration, (float*)f, (const float*)x, (const float*)v); mycudaDebugError("PlaneForceFieldCuda3t_addForce_kernel<float>");}
}

void PlaneForceFieldCuda3f1_addForce(unsigned int size, GPUPlane3f* plane, float* penetration, void* f, const void* x, const void* v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t1_addForce_kernel<float><<< grid, threads >>>(size, *plane, penetration, (CudaVec4<float>*)f, (const CudaVec4<float>*)x, (const CudaVec4<float>*)v); mycudaDebugError("PlaneForceFieldCuda3t1_addForce_kernel<float>");}
}

void PlaneForceFieldCuda3f_addDForce(unsigned int size, GPUPlane3f* plane, const float* penetration, void* df, const void* dx) //, const void* dfdx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t_addDForce_kernel<float><<< grid, threads >>>(size, *plane, penetration, (float*)df, (const float*)dx); mycudaDebugError("PlaneForceFieldCuda3t_addDForce_kernel<float>");}
}

void PlaneForceFieldCuda3f1_addDForce(unsigned int size, GPUPlane3f* plane, const float* penetration, void* df, const void* dx) //, const void* dfdx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t1_addDForce_kernel<float><<< grid, threads >>>(size, *plane, penetration, (CudaVec4<float>*)df, (const CudaVec4<float>*)dx); mycudaDebugError("PlaneForceFieldCuda3t1_addDForce_kernel<float>");}
}

#ifdef SOFA_GPU_CUDA_DOUBLE

void PlaneForceFieldCuda3d_addForce(unsigned int size, GPUPlane3d* plane, double* penetration, void* f, const void* x, const void* v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t_addForce_kernel<double><<< grid, threads >>>(size, *plane, penetration, (double*)f, (const double*)x, (const double*)v); mycudaDebugError("PlaneForceFieldCuda3t_addForce_kernel<double>");}
}

void PlaneForceFieldCuda3d1_addForce(unsigned int size, GPUPlane3d* plane, double* penetration, void* f, const void* x, const void* v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t1_addForce_kernel<double><<< grid, threads >>>(size, *plane, penetration, (CudaVec4<double>*)f, (const CudaVec4<double>*)x, (const CudaVec4<double>*)v); mycudaDebugError("PlaneForceFieldCuda3t1_addForce_kernel<double>");}
}

void PlaneForceFieldCuda3d_addDForce(unsigned int size, GPUPlane3d* plane, const double* penetration, void* df, const void* dx) //, const void* dfdx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t_addDForce_kernel<double><<< grid, threads >>>(size, *plane, penetration, (double*)df, (const double*)dx); mycudaDebugError("PlaneForceFieldCuda3t_addDForce_kernel<double>");}
}

void PlaneForceFieldCuda3d1_addDForce(unsigned int size, GPUPlane3d* plane, const double* penetration, void* df, const void* dx) //, const void* dfdx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    {PlaneForceFieldCuda3t1_addDForce_kernel<double><<< grid, threads >>>(size, *plane, penetration, (CudaVec4<double>*)df, (const CudaVec4<double>*)dx); mycudaDebugError("PlaneForceFieldCuda3t1_addDForce_kernel<double>");}
}

#endif // SOFA_GPU_CUDA_DOUBLE

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
